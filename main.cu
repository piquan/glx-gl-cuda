#include "hip/hip_runtime.h"
#include <assert.h>
#include <err.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <GL/glew.h>
// I only use one GLX extension, and I call it before my context is
// ready, so I can't use glxew for it.  Just use basic GLX and I'll
// get the extension myself instead of using glxew.
#include <GL/glx.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#define FPS 30
#define MAX_SHADER_LEN 65536
#define NREDS 1024

#define WIDTH 800
#define HEIGHT 800

static int buffer_attributes[] = {
    GLX_DRAWABLE_TYPE, GLX_WINDOW_BIT,
    GLX_RENDER_TYPE,   GLX_RGBA_BIT,
    GLX_DOUBLEBUFFER,  True,  /* Request a double-buffered color buffer with */
    GLX_RED_SIZE,      1,     /* the maximum number of bits per component    */
    GLX_GREEN_SIZE,    1, 
    GLX_BLUE_SIZE,     1,
    None
};

struct resources
{
    Display *dpy;
    GLXWindow glxWin;
    GLXContext context;

    GLuint gl_uniform_buffer;
    GLuint gl_time_uniform_loc;
    GLuint gl_vao;
    GLuint gl_vertex_buffer;
    GLuint gl_element_buffer;
    GLuint gl_program;

    hipGraphicsResource_t cuda_uniform_buffer;
};

struct reds_buffer
{
    struct {
        GLfloat red;
        GLfloat unused[3];
    } reds[NREDS];
};

static void
cuda_errchk_inner(const char* file, unsigned long line)
{
    hipError_t err = hipGetLastError();
    if (err == hipSuccess)
        return;
    const char *errstr = hipGetErrorName(err);
    fprintf(stderr, "%s:%lu: CUDA error: %s\n", file, line, errstr);
    exit(EXIT_FAILURE);
}
#define CUDA_ERRCHK() cuda_errchk_inner(__FILE__, __LINE__)

static void
gl_errchk_inner(const char* file, unsigned long line)
{
    GLenum err = glGetError();
    if (err == GL_NO_ERROR)
        return;
    const GLubyte *errstr = gluErrorString(err);
    fprintf(stderr, "%s:%lu: OpenGL error: %s\n", file, line, errstr);
    exit(EXIT_FAILURE);
}
#define GL_ERRCHK() gl_errchk_inner(__FILE__, __LINE__)

static Bool
WaitForNotify(Display *dpy, XEvent *event, XPointer arg)
{
    return (event->type == MapNotify) && (event->xmap.window == (Window)arg);
}

static void
start_gl(struct resources *rsrc)
{
    // For much of this, see:
    // https://www.khronos.org/opengl/wiki/Programming_OpenGL_in_Linux:_GLX_and_Xlib
    // https://www.khronos.org/opengl/wiki/Tutorial:_OpenGL_3.0_Context_Creation_(GLX)

    Window xWin;
    XEvent event;
    XVisualInfo *vInfo;
    XSetWindowAttributes swa;
    GLXFBConfig *fbConfigs;
    int swaMask;
    int numReturned;

    /* Open a connection to the X server */
    rsrc->dpy = XOpenDisplay(NULL);
    if (rsrc->dpy == NULL) {
        fprintf(stderr, "Unable to open a connection to the X server\n");
        exit(EXIT_FAILURE);
    }

    /* Request a suitable framebuffer configuration - try for a double 
     * buffered configuration first */
    fbConfigs = glXChooseFBConfig(rsrc->dpy, DefaultScreen(rsrc->dpy),
                                  buffer_attributes, &numReturned);

    /* Create an X colormap and window with a visual matching the first
     * returned framebuffer config */
    vInfo = glXGetVisualFromFBConfig(rsrc->dpy, fbConfigs[0]);

    swa.border_pixel = 0;
    swa.event_mask = StructureNotifyMask | ButtonPressMask | KeyPressMask;
    swa.colormap = XCreateColormap(rsrc->dpy,
                                   RootWindow(rsrc->dpy, vInfo->screen),
                                   vInfo->visual, AllocNone);

    swaMask = CWBorderPixel | CWColormap | CWEventMask;

    xWin = XCreateWindow(rsrc->dpy, RootWindow(rsrc->dpy, vInfo->screen),
                         0, 0, WIDTH, HEIGHT,
                         0, vInfo->depth, InputOutput, vInfo->visual,
                         swaMask, &swa);
    XStoreName(rsrc->dpy, xWin, "Blending CUDA and OpenGL");

    /* Create a GLX context for OpenGL rendering */
    int context_attribs[] = {
        GLX_CONTEXT_MAJOR_VERSION_ARB, 3,
        GLX_CONTEXT_MINOR_VERSION_ARB, 2,
        None
    };
    /* I can't initialize GLXEW yet because I don't have a context.
     * That means that I need to get the glXCreateContextAttribsARB
     * address myself. */
    typedef GLXContext (*glXCreateContextAttribsARBProc)
        (Display*, GLXFBConfig, GLXContext, Bool, const int*);
    glXCreateContextAttribsARBProc glXCreateContextAttribsARB =
        (glXCreateContextAttribsARBProc)glXGetProcAddressARB(
            (const GLubyte*)"glXCreateContextAttribsARB");
    rsrc->context = glXCreateContextAttribsARB(rsrc->dpy, fbConfigs[0], NULL,
                                               True, context_attribs);
    /*
      Alternative for getting an OpenGL 2 context:
    context = glXCreateNewContext(rsrc->dpy, fbConfigs[0], GLX_RGBA_TYPE,
                                  NULL, True);
    */

    /* Create a GLX window to associate the frame buffer configuration
     * with the created X window */
    rsrc->glxWin = glXCreateWindow(rsrc->dpy, fbConfigs[0], xWin, NULL);
    
    /* Map the window to the screen, and wait for it to appear */
    XMapWindow(rsrc->dpy, xWin);
    XIfEvent(rsrc->dpy, &event, WaitForNotify, (XPointer)xWin);

    /* Bind the GLX context to the Window */
    glXMakeContextCurrent(rsrc->dpy, rsrc->glxWin, rsrc->glxWin, rsrc->context);
    GL_ERRCHK();

    /* Initialize GLEW for extensions */
    glewExperimental = True;
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW error: %s\n", glewGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // GLEW's probes can leave an error in the context, so clear it.
    glGetError();
    // This is kinda redundant here, but I'm leaving it.
    GL_ERRCHK();
}

static void
start_cuda(struct resources *rsrc)
{
    // Deprecated, no longer necessary
    //cudaGLSetGLDevice(0);
    CUDA_ERRCHK();
}

static void
initialize_cuda_resources(struct resources *rsrc)
{
    hipGraphicsGLRegisterBuffer(&rsrc->cuda_uniform_buffer,
                                 rsrc->gl_uniform_buffer,
                                 hipGraphicsRegisterFlagsWriteDiscard);
    CUDA_ERRCHK();
}

static void
read_shader(const char* filename, GLchar **shader_src, GLint *shader_len)
{
    *shader_src = new char[MAX_SHADER_LEN];
    if (*shader_src == NULL)
        err(EXIT_FAILURE, "malloc");
    int fd = open(filename, O_RDONLY);
    if (fd < 0)
        err(EXIT_FAILURE, "%s", filename);
    *shader_len = read(fd, *shader_src, MAX_SHADER_LEN);
    if (*shader_len < 0)
        err(EXIT_FAILURE, "%s", filename);
    int close_err = close(fd);
    if (close_err)
        err(EXIT_FAILURE, "%s", filename);
    if (*shader_len == MAX_SHADER_LEN)
        errx(EXIT_FAILURE, "%s: Shader too long; increase MAX_SHADER_LEN",
             filename);
}

static GLuint
compile_shader(const char* filename, GLenum type)
{
    GLchar *shader_src;
    GLint shader_len;
    read_shader(filename, &shader_src, &shader_len);
    GLchar *util_src;
    GLint util_len;
    read_shader("util.glsl", &util_src, &util_len);
    
    const GLchar* shader_srcs[2];
    GLint shader_lens[2];
    shader_srcs[0] = shader_src;
    shader_lens[0] = shader_len;
    shader_srcs[1] = util_src;
    shader_lens[1] = util_len;

    GLuint shader = glCreateShader(type);
    glShaderSource(shader, 1, shader_srcs, shader_lens);
    delete[] shader_src;
    delete[] util_src;
    GL_ERRCHK();
    glCompileShader(shader);

    GLint is_compiled;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &is_compiled);
    GLint max_length = 0;
    glGetShaderiv(shader, GL_INFO_LOG_LENGTH, &max_length);
    GLchar error_log[max_length];
    glGetShaderInfoLog(shader, max_length, &max_length, &error_log[0]);
    if(is_compiled == GL_FALSE) {
        fprintf(stderr, "%s: Shader compile error:\n%s", filename, error_log);
        exit(EXIT_FAILURE);
    } else if (max_length) {
        fprintf(stderr, "%s: Shader compile messages:\n%s", filename, error_log);
    }
    
    GL_ERRCHK();
    return shader;
}

static void
initialize_gl_resources(struct resources *rsrc)
{
    /*
     * Allocate buffers
     */

    // Uniform buffer
    glGenBuffers(1, &rsrc->gl_uniform_buffer);
    // The buffer we're using would be more appropriate as a 1d
    // texture.  However, in practice, uniform buffers are more likely
    // to be used for CUDA-OpenGL interface, so I'm using that to make
    // better demo code.
    glBindBuffer(GL_UNIFORM_BUFFER, rsrc->gl_uniform_buffer);
    // This actually allocates the storage for the buffer.  The last
    // parameter determines where it will be allocated.  See also
    // https://www.khronos.org/opengl/wiki/Buffer_Object
    glBufferData(GL_UNIFORM_BUFFER, sizeof(struct reds_buffer),
                 NULL, GL_STREAM_DRAW);
    GL_ERRCHK();

    // For the following vertex-related stuff, see
    // https://www.khronos.org/opengl/wiki/Vertex_Specification

    // Vertex array object
    // This object holds all of the vertex state information that
    // we're about to set up.  We only need one for our program, so
    // we just create it and bind it.
    glGenVertexArrays(1, &rsrc->gl_vao);
    glBindVertexArray(rsrc->gl_vao);
    GL_ERRCHK();

    // Vertex buffer object
    // This holds the information that we'll pass for each vertex.
    // We'll pass the position and the blue channel.
    //
    // First, assign internal identifiers to each attribute.  We can
    // make these up; we'll assign them to actual variable names in
    // the shader later.
    const int position_attr = 0;
    const int blue_attr = 1;
    static struct vertex {
        // position is a vec3.
        GLfloat position[3];
        // blue is a unsigned byte.  We have the GPU convert it to a
        // float during the upload, so that the shader can use the
        // float-optimized hardware.  (This is a ridiculous way to handle
        // this in our case, but I'm just demonstrating float normalization
        // in VBOs.)
        GLubyte blue;
    } vertices[4] = {
        // This array shows all the vertices we'll use in this program.
        // We'll talk about the order in which they're used in the main
        // loop, but for now, note that these are conveniently arranged
        // in clockwise order starting with quadrant I.
        {{  1.0,  1.0, 0.0 }, 0},
        {{ -1.0,  1.0, 0.0 }, 255},
        {{ -1.0, -1.0, 0.0 }, 0},
        {{  1.0, -1.0, 0.0 }, 255},
    };
    // Create, bind, and populate the buffer holding this data.  We won't
    // ever change it, so use GL_STATIC_DRAW.
    glGenBuffers(1, &rsrc->gl_vertex_buffer);
    glBindBuffer(GL_ARRAY_BUFFER, rsrc->gl_vertex_buffer);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
    GL_ERRCHK();
    // Set each vertex attribute's location, so that GL knows which
    // part of the vertex to send to each variable.  (We just assign
    // these to numbers now; we'll connect those numbers to names
    // below when we compile the shaders.)
    // 
    // There are two ways to do this: one is with glVertexAttribPointer
    // (which is always available), or with glVertexAttribFormat and
    // friends (which requires the extension ARB_vertex_attrib_binding,
    // which is available in most cards supporting 3.3 and later).
    // We'll demonstrate both, although in practice you'd only use one
    // depending on your needs.
    // (You can use "0&&" etc to fiddle around with these.)
    if (GLEW_ARB_vertex_attrib_binding) {
        const int vbo_idx = 0;  // We only use one VBO; call it #0
        glBindVertexBuffer(vbo_idx, rsrc->gl_vertex_buffer, 
                           0, sizeof(struct vertex));
        glVertexAttribFormat(position_attr, 3, GL_FLOAT, GL_FALSE,
                             offsetof(struct vertex, position));
        glVertexAttribBinding(position_attr, vbo_idx);
        glVertexAttribFormat(blue_attr, 1, GL_UNSIGNED_BYTE, GL_TRUE,
                             offsetof(struct vertex, blue));
        glVertexAttribBinding(blue_attr, vbo_idx);
    } else {
        fprintf(stderr, "Huh, I'm not using ARB_vertex_attrib_binding.\n");
        glVertexAttribPointer(position_attr, 3, GL_FLOAT, GL_FALSE,
                              sizeof(struct vertex),
                              reinterpret_cast<void*>(
                                  offsetof(struct vertex, position)));
        glVertexAttribPointer(blue_attr, 1, GL_UNSIGNED_BYTE, GL_TRUE,
                              sizeof(struct vertex),
                              reinterpret_cast<void*>(
                                  offsetof(struct vertex, blue)));
    }
    glEnableVertexAttribArray(position_attr);
    glEnableVertexAttribArray(blue_attr);
    GL_ERRCHK();

    // Element Array
    //
    // This is an array that says which order we want to draw our
    // vertices in.  It's not necessary; we could put all our vertices
    // in the "vertices" array in the order desired, and use
    // glDrawArrays directly.  We're doing it this way because some of
    // our vertices are duplicates, so instead of uploading 67% more
    // vertices, we just send a list of the indices.
    //
    // Since the main loop is changing the order of the vertices,
    // we'll just set up the object, and let the main loop upload the
    // indices.
    glGenBuffers(1, &rsrc->gl_element_buffer);
    GL_ERRCHK();

    /*
     * Compile shaders
     */

    GLuint vertex_shader = compile_shader("vertex.glsl", GL_VERTEX_SHADER);
    GLuint fragment_shader =
        compile_shader("fragment.glsl", GL_FRAGMENT_SHADER);
    
    rsrc->gl_program = glCreateProgram();
    glAttachShader(rsrc->gl_program, vertex_shader);
    glAttachShader(rsrc->gl_program, fragment_shader);
    GL_ERRCHK();
    // Now we actually bind our attributes, which we assigned internal
    // numbers to earlier, to their locations in the shaders.
    glBindAttribLocation(rsrc->gl_program, position_attr, "position");
    glBindAttribLocation(rsrc->gl_program, blue_attr, "blue");
    GL_ERRCHK();
    glLinkProgram(rsrc->gl_program);
    
    GLint is_linked = 0;
    glGetProgramiv(rsrc->gl_program, GL_LINK_STATUS, &is_linked);
    GLint max_length = 0;
    glGetProgramiv(rsrc->gl_program, GL_INFO_LOG_LENGTH, &max_length);
    GLchar error_log[max_length];
    glGetProgramInfoLog(rsrc->gl_program, max_length, &max_length, error_log);
    if (is_linked == GL_FALSE) {
        fprintf(stderr, "Shader link error:\n%s", error_log);
        exit(EXIT_FAILURE);
    } else if (max_length) {
        fprintf(stderr, "Shader link messages:\n%s", error_log);
    }

    glDetachShader(rsrc->gl_program, vertex_shader);
    glDeleteShader(vertex_shader);
    glDetachShader(rsrc->gl_program, fragment_shader);
    glDeleteShader(fragment_shader);
    GL_ERRCHK();

    /*
     * Set up uniforms
     */

    // Set up the uniform block object, which is our "reds" array.
    // Get the index of the "reds" block.
    GLuint uniform_idx = glGetUniformBlockIndex(rsrc->gl_program, "reds_block");
    assert(uniform_idx != GL_INVALID_INDEX);
    // Set that up as uniform buffer #0.
    glUniformBlockBinding(rsrc->gl_program, uniform_idx, 0);
    // Bind it to our previously-created uniform buffer.
    glBindBufferBase(GL_UNIFORM_BUFFER, 0, rsrc->gl_uniform_buffer);
    GL_ERRCHK();

    // Set up the "time" uniform.
    rsrc->gl_time_uniform_loc =
        glGetUniformLocation(rsrc->gl_program, "time");
}

static __global__ void
calculate_reds_kernel(struct reds_buffer* reds_block,
                      unsigned long long time)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < NREDS) {
        float theta = sinf(float(time) / 8.0) + float(thread_id) / 64.0;
        reds_block->reds[thread_id].red = pow(sinf(theta), 2);
    }
}

static void
calculate_reds(struct resources *rsrc, unsigned long long time)
{
    // Map the uniform buffer into CUDA space so the kernel can work on it.
    hipGraphicsMapResources(1, &rsrc->cuda_uniform_buffer);
    CUDA_ERRCHK();
    // Get a CUDA-accessible pointer to the mapped buffer.
    struct reds_buffer* devptr;
    size_t devptr_size;
    hipGraphicsResourceGetMappedPointer((void**)&devptr, &devptr_size,
                                         rsrc->cuda_uniform_buffer);
    CUDA_ERRCHK();
    assert(devptr_size == sizeof(struct reds_buffer));
    // Launch the kernel.
    calculate_reds_kernel<<<16, NREDS / 16>>>(devptr, time);
    CUDA_ERRCHK();
    // Unmap the uniform buffer so it's available to OpenGL again.  (This
    // includes an implicit sync point.)
    hipGraphicsUnmapResources(1, &rsrc->cuda_uniform_buffer);
    CUDA_ERRCHK();
}

static void
load_elements(struct resources *rsrc, const GLuint* vertices,
              size_t vertices_size)
{
    glXMakeContextCurrent(rsrc->dpy, rsrc->glxWin, rsrc->glxWin, rsrc->context);
    glBindVertexArray(rsrc->gl_vao);
    glBindBuffer(GL_ARRAY_BUFFER, rsrc->gl_vertex_buffer);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, rsrc->gl_element_buffer);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, vertices_size, vertices,
                 GL_DYNAMIC_DRAW);
    GL_ERRCHK();
}

static void
draw_frame(struct resources *rsrc, GLsizei nvertices, unsigned long long time)
{
    // Activate our context, shaders, and VAO.  (Not technically
    // necessary here, since they've been activated all along, but
    // it's always prudent to refresh the context on each drawing in a
    // big program.)
    glXMakeContextCurrent(rsrc->dpy, rsrc->glxWin, rsrc->glxWin, rsrc->context);
    glUseProgram(rsrc->gl_program);
    glBindVertexArray(rsrc->gl_vao);

    // Set the time uniform.
    glUniform1f(rsrc->gl_time_uniform_loc, float(time) / FPS);
    GL_ERRCHK();

    // Start drawing
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawElements(GL_TRIANGLE_STRIP, nvertices, GL_UNSIGNED_INT, 0);
    glFlush();
    glXSwapBuffers(rsrc->dpy, rsrc->glxWin);
    GL_ERRCHK();
}

static Bool
is_quit_event(Display *dpy, XEvent *evt, XPointer arg)
{
    if (evt->type == KeyPress) {
        KeySym ks = XLookupKeysym(&evt->xkey, 0);
        return !IsModifierKey(ks);
    }
    if (evt->type == ButtonPress)
        return True;
    return False;
}

static void
check_input(struct resources *rsrc)
{
    XEvent evt;
    if (XCheckIfEvent(rsrc->dpy, &evt, is_quit_event, NULL))
        exit(EXIT_SUCCESS);
}

int
main(void)
{
    struct resources rsrc;
    start_gl(&rsrc);
    start_cuda(&rsrc);
    initialize_gl_resources(&rsrc);
    initialize_cuda_resources(&rsrc);

#if 0
    // This is an example of drawing under OpenGL 1 or 2.  This uses
    // the OpenGL built-in matrix stuff, and individual calls to
    // primitives.  The built-in matrices are not part of the core
    // profile in OpenGL 3.1 and above, but they're generally
    // available in the compatibility profile.  However, we've asked
    // for core profile, so this stuff isn't available.

    // Set up which portion of the window is being used
    glViewport(0, 0, WIDTH, HEIGHT);
    // Just set up an orthogonal system
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, 1.0f, 0, 1.0f, -1.0f, 1.0f);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glEnable(GL_DEPTH_TEST);
    glClearColor(1.0f, 1.0f, 1.0f, 1.5f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    // Here's where you'd typically put drawing commands.
    glFlush();
    glXSwapBuffers(dpy, glxWin);
    GL_ERRCHK();
#endif

    // Keep a frame counter
    unsigned long long time = 0;

    // These are the triangles that we'll render at each stage of the loop.
    // Note that we always arrange these counterclockwise, so that the
    // front of the triangle is facing us.
    static GLuint triangle_indices[][3] = {
        { 0, 1, 2 },
        { 1, 2, 3 },
        { 2, 3, 0 },
        { 3, 0, 1 }
    };
    for (int i = 0; i < 4; i++) {
        load_elements(&rsrc, triangle_indices[i], sizeof(triangle_indices[i]));
        for (int j = 0; j < FPS; j++) {
            check_input(&rsrc);
            calculate_reds(&rsrc, time);
            draw_frame(&rsrc, 3, time);
            time++;
            usleep(1000000 / FPS);
        }
    }

    // This is the triangle strip we'll render at the end of the loop.
    // Note that we need to pick the order to correctly draw the strip.
    static GLuint quad_indices[] = { 0, 1, 3, 2 };
    load_elements(&rsrc, quad_indices, sizeof(quad_indices));
    while (1) {
        check_input(&rsrc);
        calculate_reds(&rsrc, time);
        draw_frame(&rsrc, 4, time);
        time++;
        usleep(1000000 / FPS);
    }
}

/*
 * Local Variables:
 * mode: c++
 * compile-command: "/usr/local/cuda/bin/nvcc -g -O -Xcompiler=-Wall -o main -lGL -lGLU -lGLEW -lX11 main.cu && optirun ./main"
 * End:
 */
